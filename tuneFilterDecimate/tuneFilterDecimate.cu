
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h>
#include <stdlib.h>
#include <unistd.h>
#include <cstdlib>

//Data parameters
#define SHOW_ELAPSED_TIME 1

//DSP parameters
#define SAMP_FREQ 1e6
#define FREQ_SHIFT -350000

//Coefficients for FIR
#define DECIMATION_RATE 2
#define FIR_SIZE 64
const int fir_size_in_bytes = FIR_SIZE * sizeof(float);
__constant__ float fir_coef [FIR_SIZE];
float cpu_fir_coef[FIR_SIZE] = {0.0};

// Declare arrays dynamically
float *cpu_I_in_buffer;
float *cpu_I_result_buffer;
float *cpu_Q_in_buffer;
float *cpu_Q_result_buffer;

//Default values, can be overwritten by providing command line arguments
unsigned int array_size;
unsigned int array_size_in_bytes;
unsigned int num_threads = FIR_SIZE;
unsigned int num_blocks = 4096;

//Function to copy data into shared memory. Includes thread sync
__device__ 
void copy_data_to_shared(float * src, float * dst, const unsigned int tid)
{
    // Copy data
    dst[tid] = src[tid];

    // Sync threads before accessing shared memory
    __syncthreads();
}

//Function to copy data out of shared memory. Includes thread sync
__device__ 
void copy_data_from_shared(float * src, float * dst, const unsigned int tid)
{
    // Sync threads before accessing shared memory
    __syncthreads();

    // Copy data
    dst[tid] = src[tid];
}


// Custom complex multiplication kernel
__device__
void cMult(const float Ai, const float Aq, const float Bi, const float Bq, float* Ri, float* Rq)
{
    *Ri = Ai*Bi - Aq*Bq;
    *Rq = Ai*Bq + Aq*Bi;
}

//Method to quickly sum an array and put the result at the begining of the array
//http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
__device__
void sum_array(float * sdata, const unsigned int blockSize, const unsigned int tid)
{
    for (unsigned int s=blockSize/2; s>0; s>>=1) {
        if(tid < s) 
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
}

// Perform a frequency shift via complex multiply
// Parameters:
// I_in, Q_in, I_out, Q_out: 4 data buffers, all must be the same length
// n0: Used to calculate the phase of the first point of the mixing signal
// freq_shift: Frequency to shift in Hz
// Fs: sample frequency in Hz
__global__
void freq_shift(float * I_in, float * Q_in, float * I_out, float * Q_out, const unsigned int n0, const float freq_shift, const float Fs)
{
    //Who am I?
    //const unsigned int thread_id = threadIdx.x;
    //const unsigned int block_id = blockIdx.x;
    const unsigned int global_index = (blockIdx.x * blockDim.x) + threadIdx.x;

    float I_shift;
    float Q_shift;
    float theta_nopi = 2.0*freq_shift*(n0 + global_index)/Fs;   

    sincospif(theta_nopi, &Q_shift, &I_shift);
    cMult(I_in[global_index], Q_in[global_index], I_shift, Q_shift, &I_out[global_index], &Q_out[global_index]);    
}

// FIR based decimation
__global__
void decimate(float * input_buffer, float * output_buffer, const unsigned int decimation_factor)
{
    __shared__ float conv[FIR_SIZE];

    //Who am I?
    const unsigned int thread_id = threadIdx.x;
    const unsigned int block_id = blockIdx.x;
	//const unsigned int global_index = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    //Perform the convolution as a copy from global (num samples) to shared (FIR width)
    float sample = 0.0;
    int sample_index = block_id - thread_id;
    if(sample_index >= 0)
        sample = input_buffer[sample_index];    
    conv[thread_id] = sample*fir_coef[thread_id];
    __syncthreads();

    //Sum results vector using loop unrolling and shared memory
    sum_array(conv, blockDim.x, thread_id);
    
    //Decimate
    if(thread_id == 0)
    {
        if((block_id % decimation_factor) == 0)
            output_buffer[block_id / decimation_factor] = conv[0];
    }
}

// main_sub0 : Method to copy an input buffer into cuda and copy the results out
void main_sub0()
{
	// Declare pointers for GPU based params
    float *gpu_I_in_buffer;
    float *gpu_I_mixed_buffer;
    float *gpu_I_result_buffer;
    float *gpu_Q_in_buffer;
    float *gpu_Q_mixed_buffer;
    float *gpu_Q_result_buffer;

    // Allocate memory in the GPU
    hipMalloc((void **)&gpu_I_in_buffer, array_size_in_bytes);
    hipMalloc((void **)&gpu_I_mixed_buffer, array_size_in_bytes);
    hipMalloc((void **)&gpu_I_result_buffer, array_size_in_bytes);
    hipMalloc((void **)&gpu_Q_in_buffer, array_size_in_bytes);
    hipMalloc((void **)&gpu_Q_mixed_buffer, array_size_in_bytes);
    hipMalloc((void **)&gpu_Q_result_buffer, array_size_in_bytes);

    //Copy Constant data
    hipMemcpyToSymbol(HIP_SYMBOL(fir_coef), &cpu_fir_coef, fir_size_in_bytes);

#if SHOW_ELAPSED_TIME
    float ms;

    // Setup Start and Stop event
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // Start timer
    hipEventRecord(startEvent, 0);
#endif

    // Copy data from CPU to GPU
	hipMemcpy(gpu_I_in_buffer, cpu_I_in_buffer, array_size_in_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_Q_in_buffer, cpu_Q_in_buffer, array_size_in_bytes, hipMemcpyHostToDevice);

    //Run kernels
    freq_shift<<<num_blocks/32, 32>>>(gpu_I_in_buffer, gpu_Q_in_buffer, gpu_I_mixed_buffer, gpu_Q_mixed_buffer, 0, FREQ_SHIFT, SAMP_FREQ);
    decimate<<<num_blocks, num_threads>>>(gpu_I_mixed_buffer, gpu_I_result_buffer, DECIMATION_RATE);
    decimate<<<num_blocks, num_threads>>>(gpu_Q_mixed_buffer, gpu_Q_result_buffer, DECIMATION_RATE);
    
    // Copy results from GPU to CPU	
	hipMemcpy(cpu_I_result_buffer, gpu_I_result_buffer, array_size_in_bytes/DECIMATION_RATE, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_Q_result_buffer, gpu_Q_result_buffer, array_size_in_bytes/DECIMATION_RATE, hipMemcpyDeviceToHost);

#if SHOW_ELAPSED_TIME
    // Stop timer
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Elapsed Time: %f ms\n", ms);

    // Destroy timer
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
#endif

    //Destory streams
    //cudaStreamDestroy(stream1);
    //cudaStreamDestroy(stream2);

    // Free the arrays on the GPU
    hipFree(gpu_I_in_buffer);
    hipFree(gpu_I_mixed_buffer);
    hipFree(gpu_I_result_buffer);
    hipFree(gpu_Q_in_buffer);
    hipFree(gpu_Q_mixed_buffer);
    hipFree(gpu_Q_result_buffer);

}

//main : parse command line arguments and run GPU code
int main(int argc, char *argv[])
{
    // Argument parsing using getopt
    // http://www.gnu.org/software/libc/manual/html_node/Example-of-Getopt.html#Example-of-Getopt
    // -b <int> sets the number of GPU blocks
    // -t <int> sets the nubmer of GPU threads
    // -v Sets verbose flag - shows math results

    int c;
    bool showMathResults = false;
    
    while ((c = getopt (argc, argv, "b:t:v")) != -1)
    switch (c)
    {
        case 'b':
            num_blocks = atoi(optarg);
            break;
        case 't':
            num_threads = atoi(optarg);
            break;
        case 'v':
            showMathResults = true;
            break;
        default:
            printf("USAGE:\n-b <int> GPU blocks\n-t <int> GPU threads (each block)\n-v Verbose\n");
            return EXIT_SUCCESS;
    }

    printf("Blocks: %d\nThreads: %d\n", num_blocks, num_threads);

    // Calculate buffer size
    array_size = num_blocks;
    array_size_in_bytes = sizeof(float) * (array_size);

    // Allocate memory on the CPU
    cpu_I_in_buffer = new float[array_size];
    cpu_I_result_buffer = new float[array_size];
    cpu_Q_in_buffer = new float[array_size];
    cpu_Q_result_buffer = new float[array_size];

    //Load fir
    FILE * iFile;
    char fileName[100];
    sprintf(fileName, "fir_dec_%d_taps_%d.txt", DECIMATION_RATE, FIR_SIZE);
    iFile = fopen(fileName, "r");
    for(unsigned int i=0; i<FIR_SIZE; i++)
        fscanf(iFile, "%f\r\n", &cpu_fir_coef[i]);
    fclose(iFile);

    // Generate data to be processed
    float I, Q;
    iFile = fopen("inputIQ.txt", "r+");   
    for(unsigned int i=0; i<array_size; i++)
    {
        fscanf(iFile, "%f,%f\r\n", &I, &Q);
        cpu_I_in_buffer[i] = I;
        cpu_Q_in_buffer[i] = Q;
    }
    fclose(iFile);

    // Run
    main_sub0();

    // Output results
    if(showMathResults)
    {
	    for(unsigned int i = 0; i < array_size/DECIMATION_RATE; i++)
	    {
		    printf("%.5f, %.5f\n", cpu_I_result_buffer[i], cpu_Q_result_buffer[i]);
	    }
    }
    printf("\n");

    FILE * oFile;
    oFile = fopen("outputIQ.txt", "w+");
    for(unsigned int i=0; i<array_size/DECIMATION_RATE; i++)
    {
        fprintf(oFile, "%f,%f\r\n", cpu_I_result_buffer[i], cpu_Q_result_buffer[i]);
    }
    fclose(oFile);

    // Done
	return EXIT_SUCCESS;
}
